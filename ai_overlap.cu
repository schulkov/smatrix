/* exp(), sqrt() */

#include <hip/hip_runtime.h>
#include <math.h>
/* malloc(), free() */
#include <stdlib.h>
/* memset() */
#include <string.h>
#include <stdio.h>


#define MATH_PI 
/* a way to switch precision : single <-> double */
typedef double REAL_T;


#define GPU_ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



/*
    few special cases derived from CP2K general subroutine
    https://github.com/cp2k/cp2k/blob/master/src/aobasis/ai_overlap.F
*/

__host__ __device__ void overlap_primitive_ss(REAL_T *sab, REAL_T zeta, REAL_T zetb, REAL_T rab_x, REAL_T rab_y, REAL_T rab_z)
{
   REAL_T dab = sqrt(rab_x*rab_x + rab_y*rab_y + rab_z*rab_z);

   //  *** Prefactors ***
   REAL_T zetp = ((REAL_T)1.0)/(zeta+zetb);
   REAL_T pi_zetp = M_PI * zetp;
   REAL_T f0 = pi_zetp*sqrt(pi_zetp);
   REAL_T f1 = zetb*zetp;

   *sab = f0*exp(-zeta*f1*dab*dab);
}

__host__ __device__ void overlap_primitive_sp(REAL_T *sab, REAL_T zeta, REAL_T zetb, REAL_T rab_x, REAL_T rab_y, REAL_T rab_z)
{
   REAL_T dab = sqrt(rab_x*rab_x + rab_y*rab_y + rab_z*rab_z);

   //  *** Prefactors ***
   REAL_T zetp = ((REAL_T)1.0)/(zeta+zetb);
   REAL_T pi_zetp = M_PI * zetp;
   REAL_T f0 = sqrt(pi_zetp*pi_zetp*pi_zetp);
   REAL_T f1 = zetb*zetp;
   REAL_T f1_m1 = f1 - (REAL_T)1.0;
   REAL_T rbp_x = f1_m1*rab_x;
   REAL_T rbp_y = f1_m1*rab_y;
   REAL_T rbp_z = f1_m1*rab_z;

   REAL_T s0 = f0*exp(-zeta*f1*dab*dab); // [s|s]
   sab[0] = rbp_x*s0; // [s|px]
   sab[1] = rbp_y*s0; // [s|py]
   sab[2] = rbp_z*s0; // [s|pz]
}

/* the same as overlap_primitive_sp(sab, zetb, zeta, rab_x, rab_y, rab_z) due to symmetry */
__host__ __device__ void overlap_primitive_ps(REAL_T *sab, REAL_T zeta, REAL_T zetb, REAL_T rab_x, REAL_T rab_y, REAL_T rab_z)
{
   REAL_T dab = sqrt(rab_x*rab_x + rab_y*rab_y + rab_z*rab_z);

   //  *** Prefactors ***
   REAL_T zetp = ((REAL_T)1.0)/(zeta+zetb);
   REAL_T pi_zetp = M_PI * zetp;
   REAL_T f0 = sqrt(pi_zetp*pi_zetp*pi_zetp);
   REAL_T f1 = zetb*zetp;
   REAL_T rap_x = f1*rab_x;
   REAL_T rap_y = f1*rab_y;
   REAL_T rap_z = f1*rab_z;

   REAL_T s0 = f0*exp(-zeta*f1*dab*dab); // [s|s]
   sab[0] = rap_x*s0; // [px|s]
   sab[1] = rap_y*s0; // [py|s]
   sab[2] = rap_z*s0; // [pz|s]
}

__host__ __device__ void overlap_primitive_pp(REAL_T *sab, REAL_T zeta, REAL_T zetb, REAL_T rab_x, REAL_T rab_y, REAL_T rab_z)
{
   REAL_T dab = sqrt(rab_x*rab_x + rab_y*rab_y + rab_z*rab_z);

   //  *** Prefactors ***
   REAL_T zetp = ((REAL_T)1.0)/(zeta+zetb);
   REAL_T pi_zetp = M_PI * zetp;
   REAL_T f0 = sqrt(pi_zetp*pi_zetp*pi_zetp);
   REAL_T f1 = zetb*zetp;
   REAL_T f2 = ((REAL_T)0.5)*zetp;
   REAL_T rap_x = f1*rab_x;
   REAL_T rap_y = f1*rab_y;
   REAL_T rap_z = f1*rab_z;
   REAL_T rbp_x = rap_x-rab_x;
   REAL_T rbp_y = rap_y-rab_y;
   REAL_T rbp_z = rap_z-rab_z;
   REAL_T s0, s1;

   s0 = f0*exp(-zeta*f1*dab*dab); // [s|s]

   s1 = rap_x*s0; // [px|s]
   sab[0] = rbp_x*s1+f2*s0; // [px|px]
   sab[1] = rbp_y*s1; // [px|py]
   sab[2] = rbp_z*s1; // [px|pz]

   s1 = rap_y*s0; // [py|s]
   sab[3] = rbp_x*s1; // [py|px]
   sab[4] = rbp_y*s1+f2*s0; // [py|py]
   sab[5] = rbp_z*s1; // [py|pz]

   s1 = rap_z*s0; // [pz|s]
   sab[6] = rbp_x*s1; // [pz|px]
   sab[7] = rbp_y*s1; // [pz|py]
   sab[8] = rbp_z*s1+f2*s0; // [pz|pz]
}

__host__ __device__ int ncoset( int l_max ){
  int nco = 0;
  for( int l=0; l <= l_max; l++ ){
    nco += (l+1)*(l+2)/2;
  }
  return nco;
}

__host__ __device__ int coset( int lx, int ly, int lz ){
  int l = lx + ly + lz;
  int co = 1 + (l - lx)*(l - lx + 1)/2 + lz;
  return ncoset(l - 1) + co - 1 ;
}

__host__ __device__ void overlap( REAL_T *s , REAL_T zeta, REAL_T zetb, REAL_T rab_x, REAL_T rab_y, REAL_T rab_z, int la_max, int lb_max){
//       *** Calculate some prefactors ***
            REAL_T zetp = 1.0/(zeta + zetb);
            REAL_T f0 = sqrt((M_PI*zetp)*(M_PI*zetp)*(M_PI*zetp));
            REAL_T f1 = zetb*zetp;
            REAL_T f2 = 0.5*zetp;
            REAL_T f3, f4;
            REAL_T rap[3];
            REAL_T rbp[3];
            REAL_T dab = sqrt( rab_x*rab_x + rab_y*rab_y + rab_z*rab_z );

            int Na = ncoset(la_max);

//       *** Calculate the basic two-center overlap integral [s|s] ***
            s[0] = f0*exp(-zeta*f1*dab*dab); //! [s|s]
//       *** Recurrence steps: [s|s] -> [a|b] ***
            if (la_max > 0) {
//         *** Vertical recurrence steps: [s|s] -> [a|s] ***
               rap[0] = f1*rab_x ; 
               rap[1] = f1*rab_y ; 
               rap[2] = f1*rab_z ; 
//         *** [p|s] = (Pi - Ai)*[s|s]  (i = x,y,z) ***
               s[1] = rap[0]*s[0]; //! [px|s]
               s[2] = rap[1]*s[0]; //! [py|s]
               s[3] = rap[2]*s[0]; //! [pz|s]

               if (la_max > 1) {
//           *** [d|s] ***
                  f3 = f2*s[0];
                  s[4] = rap[0]*s[1] + f3;  //! [dx2|s]
                  s[5] = rap[0]*s[2] ;      //! [dxy|s]
                  s[6] = rap[0]*s[3] ;      //! [dxz|s]
                  s[7] = rap[1]*s[2] + f3 ; //! [dy2|s]
                  s[8] = rap[1]*s[3] ;      //! [dyz|s]
                  s[9] = rap[2]*s[3] + f3 ; //! [dz2|s]


                  if (la_max > 2) {
//             *** [f|s] ***
                     f3 = 2.0*f2;
                     s[10] = rap[0]*s[4] + f3*s[1] ; //! [fx3 |s]
                     s[11] = rap[0]*s[5] + f2*s[2] ; //! [fx2y|s]
                     s[12] = rap[0]*s[6] + f2*s[3] ; //! [fx2z|s]
                     s[13] = rap[0]*s[7] ;           //! [fxy2|s]
                     s[14] = rap[0]*s[8] ;           //! [fxyz|s]
                     s[15] = rap[0]*s[9] ;           //! [fxz2|s]
                     s[16] = rap[1]*s[7] + f3*s[2] ; //! [fy3 |s]
                     s[17] = rap[1]*s[8] + f2*s[3] ; //! [fy2z|s]
                     s[18] = rap[1]*s[9] ;           //! [fyz2|s]
                     s[19] = rap[2]*s[9] + f3*s[3] ; //! [fz3 |s]

                     if (la_max > 3) {
//               *** [g|s] ***
                        f4 = 3.0*f2;
                        s[20] = rap[0]*s[10] + f4*s[4] ; //! [gx4  |s]
                        s[21] = rap[0]*s[11] + f3*s[5] ; //! [gx3y |s]
                        s[22] = rap[0]*s[12] + f3*s[6] ; //! [gx3z |s]
                        s[23] = rap[0]*s[13] + f2*s[7] ; //! [gx2y2|s]
                        s[24] = rap[0]*s[14] + f2*s[8] ; //! [gx2yz|s]
                        s[25] = rap[0]*s[15] + f2*s[9] ; //! [gx2z2|s]
                        s[26] = rap[0]*s[16] ;           //! [gxy3 |s]
                        s[27] = rap[0]*s[17] ;           //! [gxy2z|s]
                        s[28] = rap[0]*s[18] ;           //! [gxyz2|s]
                        s[29] = rap[0]*s[19] ;           //! [gxz3 |s]
                        s[30] = rap[1]*s[16] + f4*s[7] ; //! [gy4  |s]
                        s[31] = rap[1]*s[17] + f3*s[8] ; //! [gy3z |s]
                        s[32] = rap[1]*s[18] + f2*s[9] ; //! [gy2z2|s]
                        s[33] = rap[1]*s[19] ;           //! [gyz3 |s]
                        s[34] = rap[2]*s[19] + f4*s[9] ; //! [gz4  |s]

//               *** [a|s] = (Pi - Ai)*[a-1i|s] + f2*Ni(a-1i)*[a-2i|s] ***
                        for ( int la = 5; la <= la_max ; la++) { // DO la = 5, la_max

//                 *** Increase the angular momentum component z of a ***
                           s[coset(0, 0, la)] = rap[2]*s[coset(0, 0, la - 1)] + f2*(la - 1)*s[coset(0, 0, la - 2)];

//                 *** Increase the angular momentum component y of a ***
                           int az = la - 1;
                           s[coset(0, 1, az)] = rap[1]*s[coset(0, 0, az)] ;
                           for ( int ay = 2 ; ay <= la ; ay++ ){ // DO ay = 2, la
                              az = la - ay ;
                              s[coset(0, ay, az)] = rap[1]*s[coset(0, ay - 1, az)] + f2*(ay - 1)*s[coset(0, ay - 2, az)];
                           } // END DO

//                 *** Increase the angular momentum component x of a ***
                           for ( int ay=0 ; ay <= la-1 ; ay++ ){ // DO ay = 0, la - 1
                              az = la - 1 - ay ;
                              s[coset(1, ay, az)] = rap[0]*s[coset(0, ay, az)];
                           } // END DO
                           for ( int ax=2 ; ax <= la ; ax++ ) { // DO ax = 2, la
                              f3 = f2*(ax - 1) ;
                              for (int ay=0; ay <= la-ax ; ay++ ) { // DO ay = 0, la - ax
                                 az = la - ax - ay ;
                                 s[coset(ax, ay, az)] = rap[0]*s[coset(ax - 1, ay, az)] + f3*s[coset(ax - 2, ay, az)] ;
                              } // END DO
                           } // END DO
                        } // END DO la = 5, la_max
                     } // END IF la_max > 3
                  } // END IF la_max > 2
               } // END IF la_max > 1

//         *** Recurrence steps: [a|s] -> [a|b] ***

               if (lb_max > 0) {

// !           *** Horizontal recurrence steps ***

                  rbp[0] = rap[0] - rab_x ;
                  rbp[1] = rap[1] - rab_y ;
                  rbp[2] = rap[2] - rab_z ;

// !           *** [a|p] = [a+1i|s] - (Bi - Ai)*[a|s] ***
                  int la_start = 0;

                  for ( int la=la_start ; la <= (la_max-1); la++) { // DO la = la_start, la_max - 1
                     for( int ax=0; ax <= la; ax++ ){ // DO ax = 0, la
                        for( int ay=0; ay <= (la-ax); ay++ ){ // DO ay = 0, la - ax
                           int az = la - ax - ay ;
                           int coa = coset(ax, ay, az) ;
                           int coapx = coset(ax + 1, ay, az) ;
                           int coapy = coset(ax, ay + 1, az) ;
                           int coapz = coset(ax, ay, az + 1) ;
                           s[1*Na+ coa] = s[coapx] - rab_x*s[coa] ;
                           s[2*Na+ coa] = s[coapy] - rab_y*s[coa] ;
                           s[3*Na+ coa] = s[coapz] - rab_z*s[coa] ;
                        }
                     }
                  }

// !           *** Vertical recurrence step ***

// !           *** [a|p] = (Pi - Bi)*[a|s] + f2*Ni(a)*[a-1i|s] ***

                  for ( int ax=0; ax <= la_max; ax++ ){ // DO ax = 0, la_max
                     REAL_T fax = f2*ax ;
                     for ( int ay=0; ay <= la_max-ax ; ay++ ){ // DO ay = 0, la_max - ax
                        REAL_T fay = f2*ay;
                        int az = la_max - ax - ay;
                        REAL_T faz = f2*az;
                        int coa = coset(ax, ay, az);
                        int coamx = coset(ax - 1, ay, az);
                        int coamy = coset(ax, ay - 1, az);
                        int coamz = coset(ax, ay, az - 1);
                        s[1*Na+ coa] = rbp[0]*s[coa] + fax*s[coamx];
                        s[2*Na+ coa] = rbp[1]*s[coa] + fay*s[coamy];
                        s[3*Na+ coa] = rbp[2]*s[coa] + faz*s[coamz];

                     }
                  }

// !           *** Recurrence steps: [a|p] -> [a|b] ***

                  for ( int lb=2; lb <= lb_max ; lb++ ){ // DO lb = 2, lb_max

// !             *** Horizontal recurrence steps ***

// !             *** [a|b] = [a+1i|b-1i] - (Bi - Ai)*[a|b-1i] ***

                     la_start = 0;
                     for( int la=la_start; la <= la_max-1; la++) { // DO la = la_start, la_max - 1
                        for( int ax=0; ax <= la; ax++ ){ // DO ax = 0, la
                           for( int ay=0; ay <= la-ax; ay++ ){
                              int az = la - ax - ay;
                              int coa = coset(ax, ay, az);
                              int coapx = coset(ax + 1, ay, az);
                              int coapy = coset(ax, ay + 1, az);
                              int coapz = coset(ax, ay, az + 1);

// !                   *** Shift of angular momentum component z from a to b ***

                              int cob = coset(0, 0, lb);
                              int cobmz = coset(0, 0, lb - 1);
                              s[cob*Na+ coa] = s[cobmz*Na+ coapz] - rab_z*s[cobmz*Na+ coa];

// !                   *** Shift of angular momentum component y from a to b ***

                              for ( int by=1; by < lb; by++){ // DO by = 1, lb
                                 int bz = lb - by;
                                 int cob = coset(0, by, bz);
                                 int cobmy = coset(0, by - 1, bz);
                                 s[cob*Na+ coa] = s[cobmy*Na+ coapy] - rab_y*s[cobmy*Na+ coa];
                              }

// !                   *** Shift of angular momentum component x from a to b ***

                              for ( int bx=1; bx <= lb; bx++){ // DO bx = 1, lb
                                 for( int by=0; by <= lb-bx; by++) { // DO by = 0, lb - bx
                                    int bz = lb - bx - by;
                                    int cob = coset(bx, by, bz);
                                    int cobmx = coset(bx - 1, by, bz);
                                    s[cob*Na+ coa] = s[cobmx*Na+ coapx] - rab_x*s[cobmx*Na+ coa];
                                 }
                              }

                           }
                        }
                     }

// !             *** Vertical recurrence step ***

// !             *** [a|b] = (Pi - Bi)*[a|b-1i] + f2*Ni(a)*[a-1i|b-1i] + ***
// !             ***         f2*Ni(b-1i)*[a|b-2i]                        ***

                     for( int ax=0; ax <= la_max; ax++ ){ // DO ax = 0, la_max
                        REAL_T fax = f2*ax;
                        for( int ay=0; ay<=la_max-ax; ay++){ // DO ay = 0, la_max - ax
                           REAL_T fay = f2*ay;
                           int az = la_max - ax - ay;
                           REAL_T faz = f2*az;
                           int coa = coset(ax, ay, az);
                           int coamx = coset(ax - 1, ay, az);
                           int coamy = coset(ax, ay - 1, az);
                           int coamz = coset(ax, ay, az - 1);

// !                 *** Increase the angular momentum component z of b ***

                           REAL_T f3 = f2*(lb - 1);
                           int cob = coset(0, 0, lb);
                           int cobmz = coset(0, 0, lb - 1);
                           int cobm2z = coset(0, 0, lb - 2);
                           s[cob*Na+ coa] = rbp[2]*s[cobmz*Na+ coa] + faz*s[cobmz*Na+ coamz] + f3*s[cobm2z*Na+ coa];

// !                 *** Increase the angular momentum component y of b ***

                           int bz = lb - 1;
                           cob = coset(0, 1, bz);
                           int cobmy = coset(0, 0, bz);
                           s[cob*Na+ coa] = rbp[1]*s[cobmy*Na+ coa] + fay*s[cobmy*Na+ coamy];

                           for( int by=2; by <= lb; by++) { // DO by = 2, lb
                              int bz = lb - by;
                              REAL_T f3 = f2*(by - 1);
                              cob = coset(0, by, bz);
                              cobmy = coset(0, by - 1, bz);
                              int cobm2y = coset(0, by - 2, bz);
                              s[cob*Na+ coa] = rbp[1]*s[cobmy*Na+ coa] + fay*s[cobmy*Na+ coamy] + f3*s[cobm2y*Na+ coa];
                           }

// !                 *** Increase the angular momentum component x of b ***

                           for( int by=0; by <= lb-1; by++ ){ // DO by = 0, lb - 1
                              bz = lb - 1 - by;
                              cob = coset(1, by, bz);
                              int cobmx = coset(0, by, bz);
                              s[cob*Na+ coa] = rbp[0]*s[cobmx*Na+ coa] + fax*s[cobmx*Na+ coamx];
                           }

                           for ( int bx=2; bx <= lb; bx++ ){ // DO bx = 2, lb
                              REAL_T f3 = f2*(bx - 1);
                              for( int by=0; by <= lb-bx; by++) { // DO by = 0, lb - bx
                                 bz = lb - bx - by;
                                 cob = coset(bx, by, bz);
                                 int cobmx = coset(bx - 1, by, bz);
                                 int cobm2x = coset(bx - 2, by, bz);
                                 s[cob*Na+ coa] = rbp[0]*s[cobmx*Na+ coa] + fax*s[cobmx*Na+ coamx] + f3*s[cobm2x*Na+ coa];
                              }
                           }
                        } // for( int ay=0; ay<=la_max-ax; ay++)
                     } // for( int ax=0; ax <= la_max; ax++ )
                  }//for ( int lb=2; lb <= lb_max ; lb++ ){
               } // END if (lb_max > 0) {
            } else { // this is the else in 'if (la_max > 0) { } else {}'

               if( lb_max>0) { // IF (lb_max > 0) THEN

// !           *** Vertical recurrence steps: [s|s] -> [s|b] ***

                  rbp[0] = (f1 - 1.0)*rab_x;
                  rbp[1] = (f1 - 1.0)*rab_y;
                  rbp[2] = (f1 - 1.0)*rab_z;

// !           *** [s|p] = (Pi - Bi)*[s|s] ***

                  s[1*Na+0] = rbp[0]*s[0]; // ! [s|px]
                  s[2*Na+0] = rbp[1]*s[0]; // ! [s|py]
                  s[3*Na+0] = rbp[2]*s[0]; // ! [s|pz]

                  if (lb_max > 1) {

// !             *** [s|d] ***

                     f3 = f2*s[0];

                     s[4*Na+0] = rbp[0]*s[1*Na+0] + f3; // ! [s|dx2]
                     s[5*Na+0] = rbp[0]*s[2*Na+0]; // ! [s|dxy]
                     s[6*Na+0] = rbp[0]*s[3*Na+0]; // ! [s|dxz]
                     s[7*Na+0] = rbp[1]*s[2*Na+0] + f3; // ! [s|dy2]
                     s[8*Na+0] = rbp[1]*s[3*Na+0]; // ! [s|dyz]
                     s[9*Na+0] = rbp[2]*s[3*Na+0] + f3; // ! [s|dz2]

// !             *** [s|b] = (Pi - Bi)*[s|b-1i] + f2*Ni(b-1i)*[s|b-2i] ***

                     for( int lb=3; lb <= lb_max; lb++ ){ // DO lb = 3, lb_max

// !               *** Increase the angular momentum component z of b ***

                        s[coset(0, 0, lb)*Na+ 0] = rbp[2]*s[coset(0, 0, lb - 1)*Na+ 0] + f2*(lb - 1)*s[coset(0, 0, lb - 2)*Na+ 0];

// !               *** Increase the angular momentum component y of b ***

                        int bz = lb - 1;
                        s[coset(0, 1, bz)*Na+ 0] = rbp[1]*s[coset(0, 0, bz)*Na+ 0];
                        for( int by=2; by <= lb; lb++ ){ // DO by = 2, lb
                           bz = lb - by;
                           s[coset(0, by, bz)*Na+ 0] = rbp[1]*s[coset(0, by - 1, bz)*Na+ 0] + f2*(by - 1)*s[coset(0, by - 2, bz)*Na+ 0];
                        }

// !               *** Increase the angular momentum component x of b ***

                        for( int by=0; by <= lb-1; by++ ){ // DO by = 0, lb - 1
                           bz = lb - 1 - by;
                           s[coset(1, by, bz)*Na+ 0] = rbp[0]*s[coset(0, by, bz)*Na+ 0];
                        }
                        for( int bx=2; bx <= lb; bx++ ){ // DO bx = 2, lb
                           f3 = f2*(bx - 1);
                           for( int by=0; by <= lb-bx; by++){ // DO by = 0, lb - bx
                              bz = lb - bx - by;
                              s[coset(bx, by, bz)*Na+ 0] = rbp[0]*s[coset(bx - 1, by, bz)*Na+ 0] + f3*s[coset(bx - 2, by, bz)*Na+ 0];
                           }
                        }

                     } // for( int lb=3; lb <= lb_max; lb++ ){

                  } // if (lb_max > 1)

               } // if( lb_max>0) {

            } // if (la_max > 0) { } else {}


  	


}


__host__ __device__ inline unsigned int get_nco(int l)
{
   unsigned int nco = 0;

   if (l >= 0) nco = (l+1)*(l+2)/2;
   return nco;
}


__global__ void overlap_ab_cgf_kernel(
    REAL_T* sab_dev, REAL_T* sab_pgf_dev, REAL_T* gcc_a_dev, REAL_T* gcc_b_dev, REAL_T* zet_a_dev, REAL_T* zet_b_dev,
    int la_set, int lb_set, unsigned int ncoa, unsigned int ncob, REAL_T rab_x, REAL_T rab_y, REAL_T rab_z )
{
   unsigned int ipgf_a = threadIdx.x ;
   unsigned int ipgf_b = threadIdx.y ;
   int npgf_a = blockDim.x;
   int npgf_b = blockDim.y;
   REAL_T gccSgcc_ab;
   /*
      For each pair of primitives ipgf_a,ipgf_b, fill S_ab_pgf with the correct polynomial[r,e-mr2]
      Once that is done, accumulate the S_ab = <c_a|S_ab_pgf|c_b> on the ncoa,ncob matrix
      With  S_ab_pgf = product between primitive gaussian functions
            S_ab     = product between contracted gaussian functions
            c_a      = contraction coefficients of a
   */
   if (la_set == 0 && lb_set == 0) {
      overlap_primitive_ss(&sab_pgf_dev[(ipgf_a*npgf_b+ipgf_b)*ncoa*ncob], zet_a_dev[ipgf_a], zet_b_dev[ipgf_b], rab_x, rab_y, rab_z);
   } else if (la_set == 0 && lb_set == 1) {
      overlap_primitive_sp(&sab_pgf_dev[(ipgf_a*npgf_b+ipgf_b)*ncoa*ncob], zet_a_dev[ipgf_a], zet_b_dev[ipgf_b], rab_x, rab_y, rab_z);
   } else if (la_set == 1 && lb_set == 0) {
      overlap_primitive_ps(&sab_pgf_dev[(ipgf_a*npgf_b+ipgf_b)*ncoa*ncob], zet_a_dev[ipgf_a], zet_b_dev[ipgf_b], rab_x, rab_y, rab_z);
   } else if (la_set == 1 && lb_set == 1) {
      overlap_primitive_pp(&sab_pgf_dev[(ipgf_a*npgf_b+ipgf_b)*ncoa*ncob], zet_a_dev[ipgf_a], zet_b_dev[ipgf_b], rab_x, rab_y, rab_z);
   }

   for (unsigned int icob = 0; icob < ncob; ++icob) {
      for (unsigned int icoa = 0; icoa < ncoa; ++icoa) {
         gccSgcc_ab = sab_pgf_dev[(ipgf_a*npgf_b+ipgf_b)*ncoa*ncob+icob*ncoa+icoa] * 
                         gcc_a_dev[icoa*npgf_a+ipgf_a] * gcc_b_dev[icob*npgf_b+ipgf_b];
         atomicAdd_block(&sab_dev[icob*ncoa+icoa], gccSgcc_ab);
      }
  }
}



extern "C" {
/*
   overlap integral v1, unoptmized
   sab : overlap matrix element over contracted Gaussian functions
   la_set, lb_set : angular momenta
   npgf_a, npgf_b : number of primitive Gaussian functions in contracted sets
   zet_a(1:npgf_a), zet_b(1:npgf_b) : Gaussian exponents
   gcc_a(1:npgf_a, 1:ncoa), gcc_b(1:npgf_b, 1:ncob) : Gaussian contracted coefficients for each primitive function and each Cartesian component

   Unlike Fortran, arrays' indicies in C start from 0.
*/
void overlap_ab_cgf_gpu_legacy(
   REAL_T *sab, int la_set, int npgf_a, const REAL_T *zet_a, const REAL_T *gcc_a,
   int lb_set, int npgf_b, const REAL_T *zet_b, const REAL_T *gcc_b, REAL_T rab_x, REAL_T rab_y, REAL_T rab_z)
{
   unsigned int ncoa = get_nco(la_set);
   unsigned int ncob = get_nco(lb_set);
   REAL_T *sab_pgf_dev = NULL;
   REAL_T *sab_dev = NULL;
   REAL_T *zet_a_dev = NULL;
   REAL_T *zet_b_dev = NULL;
   REAL_T *gcc_a_dev = NULL;
   REAL_T *gcc_b_dev = NULL;

   GPU_ERROR_CHECK(hipMalloc( (void**) &sab_pgf_dev, npgf_a*npgf_b*ncoa*ncob*sizeof(REAL_T) ));
   GPU_ERROR_CHECK(hipMalloc( (void**) &sab_dev, ncoa*ncob*sizeof(REAL_T) ));
   GPU_ERROR_CHECK(hipMalloc( (void**) &zet_a_dev, npgf_a*sizeof(REAL_T) ));
   GPU_ERROR_CHECK(hipMalloc( (void**) &zet_b_dev, npgf_b*sizeof(REAL_T) ));
   GPU_ERROR_CHECK(hipMalloc( (void**) &gcc_a_dev, ncoa*npgf_a*sizeof(REAL_T) ));
   GPU_ERROR_CHECK(hipMalloc( (void**) &gcc_b_dev, ncob*npgf_b*sizeof(REAL_T) ));
   
   GPU_ERROR_CHECK(hipMemcpy( zet_a_dev, zet_a, npgf_a*sizeof(REAL_T), hipMemcpyHostToDevice ));
   GPU_ERROR_CHECK(hipMemcpy( zet_b_dev, zet_b, npgf_b*sizeof(REAL_T), hipMemcpyHostToDevice ));
   GPU_ERROR_CHECK(hipMemcpy( gcc_a_dev, gcc_a, ncoa*npgf_a*sizeof(REAL_T), hipMemcpyHostToDevice ));
   GPU_ERROR_CHECK(hipMemcpy( gcc_b_dev, gcc_b, ncob*npgf_b*sizeof(REAL_T), hipMemcpyHostToDevice ));
   GPU_ERROR_CHECK(hipMemset( sab_dev, 0, ncoa*ncob*sizeof(REAL_T)));

   dim3 npgf_ab(npgf_a, npgf_b);
//   printf("A %d %d %d \n", npgf_a*npgf_b, ncoa, ncob);
   overlap_ab_cgf_kernel<<<1, npgf_ab >>>(
         sab_dev, sab_pgf_dev, gcc_a_dev, gcc_b_dev, zet_a_dev, zet_b_dev,
         la_set, lb_set, ncoa, ncob, rab_x, rab_y, rab_z );
   GPU_ERROR_CHECK(hipGetLastError() );
   GPU_ERROR_CHECK(hipMemcpy( sab, sab_dev, ncoa*ncob*sizeof(REAL_T), hipMemcpyDeviceToHost ));
   GPU_ERROR_CHECK(hipFree(zet_a_dev));
   GPU_ERROR_CHECK(hipFree(zet_b_dev));
   GPU_ERROR_CHECK(hipFree(gcc_a_dev));
   GPU_ERROR_CHECK(hipFree(gcc_b_dev));
   GPU_ERROR_CHECK(hipFree(sab_pgf_dev));
   GPU_ERROR_CHECK(hipFree(sab_dev));
}

void overlap_ab_cgf(REAL_T *sab, int la_set, int npgf_a, const REAL_T *zet_a, const REAL_T *gcc_a, int lb_set, int npgf_b, const REAL_T *zet_b,
                    const REAL_T *gcc_b, REAL_T rab_x, REAL_T rab_y, REAL_T rab_z)
{
   unsigned int ncoa = get_nco(la_set);
   unsigned int ncob = get_nco(lb_set);
   REAL_T *sab_pgf = NULL;

   sab_pgf = (REAL_T*) malloc(ncoa*ncob*sizeof(*sab_pgf));
   if (sab_pgf == NULL) return;

   memset(sab, 0, ncoa*ncob*sizeof(*sab));

   for ( int ipgf_b = 0; ipgf_b < npgf_b; ++ipgf_b) {
      for ( int ipgf_a = 0; ipgf_a < npgf_a; ++ipgf_a) {
          if (la_set == 0 && lb_set == 0) {
             overlap_primitive_ss(sab_pgf, zet_a[ipgf_a], zet_b[ipgf_b], rab_x, rab_y, rab_z);
          } else if (la_set == 0 && lb_set == 1) {
             overlap_primitive_sp(sab_pgf, zet_a[ipgf_a], zet_b[ipgf_b], rab_x, rab_y, rab_z);
          } else if (la_set == 1 && lb_set == 0) {
             overlap_primitive_ps(sab_pgf, zet_a[ipgf_a], zet_b[ipgf_b], rab_x, rab_y, rab_z);
          } else if (la_set == 1 && lb_set == 1) {
             overlap_primitive_pp(sab_pgf, zet_a[ipgf_a], zet_b[ipgf_b], rab_x, rab_y, rab_z);
          }

          for (unsigned int icob = 0; icob < ncob; ++icob) {
             for (unsigned int icoa = 0; icoa < ncoa; ++icoa) {
                sab[icob*ncoa+icoa] += sab_pgf[icob*ncoa+icoa] * gcc_a[icoa*npgf_a+ipgf_a] * gcc_b[icob*npgf_b+ipgf_b];
             }
          }
      }
   }

   free(sab_pgf);
}

#define PAL_SLOTS 3
#define BAS_SLOTS 8
#define BAS_OFFSET_L 1
#define BAS_OFFSET_NPGF 2
#define BAS_OFFSET_Z 5
#define BAS_OFFSET_C 6
#define BAS_OFFSET_R 7

//   call compute_s ( list_ijd, atm, bas, env, s_sparse )
//  dim3 max_npgf_ab(max_npgf_col, mx_npgf_row)
//__global__ void compute_s_gpu<<< n_pairs, max_npgf_ab>>> ( int* list_ijd_dev, int* bas_dev, double* env_dev, double* s_sparse_dev )

__global__ void compute_s_gpu_kernel ( int* list_ijd_dev, int* bas_dev, double* env_dev, double* s_sparse_dev )
{
   int ijd_idx = blockIdx.x * PAL_SLOTS;
   int i = ( list_ijd_dev[ ijd_idx + 0 ] - 1 ) * BAS_SLOTS;
   int j = ( list_ijd_dev[ ijd_idx + 1 ] - 1 ) * BAS_SLOTS;
   int s_offset = list_ijd_dev[ ijd_idx + 2 ] - 1 ; // might be pushed to after the if, but it is more elegant here
   int ipgf_a = threadIdx.x;
   int ipgf_b = threadIdx.y;
   int npgf_a = bas_dev[i+BAS_OFFSET_NPGF];
   int npgf_b = bas_dev[j+BAS_OFFSET_NPGF];
   // We size the block to accomodate the largest contractionso smaller contractions only use a subset of the threads
   // so smaller contractions only use a subset of the threads
   // worse case is a contraction with high angular moment and a single coefficient
   // in which case one thread is doing all L calculations
   if ( (ipgf_a<npgf_a) and(ipgf_b<npgf_b)) {
      int la = bas_dev[i+BAS_OFFSET_L];
      int lb = bas_dev[j+BAS_OFFSET_L];
      int ncoa = get_nco(la);
      int ncob = get_nco(lb);
      double zet_a = env_dev[ bas_dev[i+BAS_OFFSET_Z] + ipgf_a ];
      double zet_b = env_dev[ bas_dev[j+BAS_OFFSET_Z] + ipgf_b ];
      double* gcc_a = &env_dev[ bas_dev[i+BAS_OFFSET_C] ];
      double* gcc_b = &env_dev[ bas_dev[j+BAS_OFFSET_C] ];
      double ra_x = env_dev[ bas_dev[i+BAS_OFFSET_R] + 0 ];
      double ra_y = env_dev[ bas_dev[i+BAS_OFFSET_R] + 1 ];
      double ra_z = env_dev[ bas_dev[i+BAS_OFFSET_R] + 2 ];
      double rb_x = env_dev[ bas_dev[j+BAS_OFFSET_R] + 0 ];
      double rb_y = env_dev[ bas_dev[j+BAS_OFFSET_R] + 1 ];
      double rb_z = env_dev[ bas_dev[j+BAS_OFFSET_R] + 2 ];
      double rab_x = ra_x - rb_x;
      double rab_y = ra_y - rb_y;
      double rab_z = ra_z - rb_z;
//      double sab_pgf_spher[9]; // nsoa*nsob]; // if L = 6, this is ((2*6+1)**2 = 169 doubles per thread. Also, this needs to be constant ( at compile time ?)
      double sab_pgf[9]; // ncoa*ncob]; // if L = 6, this is ((6+1)*(6+2)/2)**2 = 784 doubles per thread. Not great. Also, this needs to be constant ( at compile time ?)
      double s[16]; // ncoseta*ncosetb]; // if L = 6, this is [sum from 1 to 6 of ((l+1)*(l+2)/2)]**2 = 7056 doubles per thread. Not great. At all. Also, this needs to be constant ( at compile time ?)
      double cSc_ab;
      sab_pgf[0] = 0.0 ;
      sab_pgf[1] = 0.0 ;
      sab_pgf[2] = 0.0 ;
      sab_pgf[3] = 0.0 ;
      sab_pgf[4] = 0.0 ;
      sab_pgf[5] = 0.0 ;
      sab_pgf[6] = 0.0 ;
      sab_pgf[7] = 0.0 ;
      sab_pgf[8] = 0.0 ;

      //
      // Compute the gaussian integrals and saves them in sab_pgf
/*      if (la == 0 && lb == 0) {
         overlap_primitive_ss(&sab_pgf[0], zet_a, zet_b, rab_x, rab_y, rab_z);
      } else if (la == 0 && lb == 1) {
         overlap_primitive_sp(sab_pgf, zet_a, zet_b, rab_x, rab_y, rab_z);
      } else if (la == 1 && lb == 0) {
         overlap_primitive_ps(sab_pgf, zet_a, zet_b, rab_x, rab_y, rab_z);
      } else if (la == 1 && lb == 1) {
         overlap_primitive_pp(sab_pgf, zet_a, zet_b, rab_x, rab_y, rab_z);
      }
*/

      overlap( s, zet_a, zet_b, rab_x, rab_y, rab_z, la, lb );

      if (la == 0 && lb == 0) {
         sab_pgf[0] = s[0];
      } else if (la == 0 && lb == 1) {
         sab_pgf[0] = s[1];
         sab_pgf[1] = s[2];
         sab_pgf[2] = s[3];
      } else if (la == 1 && lb == 0) {
         sab_pgf[0] = s[1];
         sab_pgf[1] = s[2];
         sab_pgf[2] = s[3];
      } else if (la == 1 && lb == 1) {
         sab_pgf[0] = s[5];
         sab_pgf[1] = s[6];
         sab_pgf[2] = s[7];
         sab_pgf[3] = s[9];
         sab_pgf[4] = s[10];
         sab_pgf[5] = s[11];
         sab_pgf[6] = s[13];
         sab_pgf[7] = s[14];
         sab_pgf[8] = s[15];
      }
//      printf("BlockIdx %d ThreadIdx %d ThreadIdy %d s_offset %d la %d lb %d  %e %e %e %e %e %e %e %e %e \n ", 
//              blockIdx.x, threadIdx.x, threadIdx.y, s_offset,   la,   lb,
//              sab_pgf[0],sab_pgf[1],sab_pgf[2],sab_pgf[3],sab_pgf[4],sab_pgf[5],sab_pgf[6],sab_pgf[7],sab_pgf[8]);

      // Contract the gaussian integrals to the different products between basis set functions
      for (unsigned int icob = 0; icob < ncob; ++icob) {
         for (unsigned int icoa = 0; icoa < ncoa; ++icoa) {
            cSc_ab = sab_pgf[icob*ncoa+icoa] *  gcc_a[icoa*npgf_a+ipgf_a] * gcc_b[icob*npgf_b+ipgf_b];
            // Thanks to s_offset, writes to sab_dev from different blocks will never overlap
            atomicAdd_block(&s_sparse_dev[s_offset + icob*ncoa+icoa ], cSc_ab);
         }
      }
   }
}

void compute_s_gpu ( int* list_ijd, int* bas, REAL_T* env, REAL_T* s_sparse,
                     int n_pairs,   int nbas, int env_size, int s_sparse_size,
                     int max_npgf_col, int max_npgf_row )
{
   int* list_ijd_dev = NULL;
   int* bas_dev = NULL;
   REAL_T* env_dev = NULL;
   REAL_T* s_sparse_dev = NULL;
   dim3 max_npgf_ab(max_npgf_col, max_npgf_row);

   // copy list of pairs and enviroment to gpu
   GPU_ERROR_CHECK(hipMalloc( (void**) &list_ijd_dev, n_pairs * PAL_SLOTS * sizeof(int) ));
   GPU_ERROR_CHECK(hipMalloc( (void**) &bas_dev, nbas * BAS_SLOTS * sizeof(int) ));
   GPU_ERROR_CHECK(hipMalloc( (void**) &env_dev, env_size * sizeof(REAL_T) ));
   GPU_ERROR_CHECK(hipMalloc( (void**) &s_sparse_dev, s_sparse_size * sizeof(REAL_T) ));

   GPU_ERROR_CHECK(hipMemcpy( list_ijd_dev, list_ijd, n_pairs * PAL_SLOTS * sizeof(int), hipMemcpyHostToDevice ));
   GPU_ERROR_CHECK(hipMemcpy( bas_dev, bas, nbas * BAS_SLOTS * sizeof(int), hipMemcpyHostToDevice ));
   GPU_ERROR_CHECK(hipMemcpy( env_dev, env, env_size * sizeof(REAL_T), hipMemcpyHostToDevice ));
   GPU_ERROR_CHECK(hipMemset( s_sparse_dev, 0.0, s_sparse_size*sizeof(REAL_T)));
   // work
   compute_s_gpu_kernel<<< n_pairs, max_npgf_ab>>> ( list_ijd_dev, bas_dev, env_dev, s_sparse_dev );
   GPU_ERROR_CHECK(hipGetLastError() );
   // copy back to ram and free memory
   GPU_ERROR_CHECK(hipMemcpy( s_sparse, s_sparse_dev, s_sparse_size * sizeof(REAL_T), hipMemcpyDeviceToHost ));
   GPU_ERROR_CHECK(hipFree(list_ijd_dev));
   GPU_ERROR_CHECK(hipFree(bas_dev));
   GPU_ERROR_CHECK(hipFree(env_dev));
   GPU_ERROR_CHECK(hipFree(s_sparse_dev));
}


void norm_cgf_gto(int l_set, int npgf, const REAL_T *zet, const REAL_T *gcc, REAL_T *gcc_total)
{

   unsigned int nco = get_nco(l_set);
   REAL_T *sab = NULL;
   REAL_T zero = (REAL_T)0.0;
   REAL_T norm;

   // sab(1:nco, 1:nco, 1:npgf, 1:npgf)
   sab = (REAL_T*) malloc(nco*nco*npgf*npgf*sizeof(*sab));
   if (sab == NULL) return;

   for (int ipgf = 0; ipgf < npgf; ++ipgf) {
      for (int jpgf = 0; jpgf < npgf; ++jpgf) {
          if (l_set == 0) {
             // sab(:, :, jpgf, ipgf)
             overlap_primitive_ss(sab+(ipgf*npgf+jpgf)*nco*nco, zet[jpgf], zet[ipgf], zero, zero, zero);
          } else if (l_set == 1) {
             overlap_primitive_pp(sab+(ipgf*npgf+jpgf)*nco*nco, zet[jpgf], zet[ipgf], zero, zero, zero);
          }
      }
   }

   for (unsigned int ico = 0; ico < nco; ++ico) {
      for (int ipgf = 0; ipgf < npgf; ++ipgf) {
         // sab(ico, ico, ipgf, ipgf)
         gcc_total[ico*npgf+ipgf] = gcc[ipgf] / sqrt(sab[((ipgf*npgf+ipgf)*nco+ico)*nco+ico]);
      }

      norm = (REAL_T)0.0;
      for ( int ipgf = 0; ipgf < npgf; ++ipgf) {
         for ( int jpgf = 0; jpgf < npgf; ++jpgf) {
             // sab(ico, ico, jpgf, ipgf)
             norm += sab[((ipgf*npgf+jpgf)*nco+ico)*nco+ico] * gcc_total[ico*npgf+jpgf] * gcc_total[ico*npgf+ipgf];
         }
      }

      norm = (REAL_T)1.0 / sqrt(norm);

      for ( int ipgf = 0; ipgf < npgf; ++ipgf) {
         gcc_total[ico*npgf+ipgf] = gcc_total[ico*npgf+ipgf] * norm;
      }
   }

   free(sab);
}


} // end of extern C
